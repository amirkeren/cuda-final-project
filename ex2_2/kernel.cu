#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include "resizable_memory.h"
#include "measurement_class.h"
#include "kernel.h"

__global__ void fill_vector(unsigned int n, float *vec, float value)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < n)
		vec[i] = value;
}

template <typename data_type>
__global__ void ell_spmv_kernel(
	unsigned int n_rows,
	unsigned int elements_in_rows,
	const unsigned int *col_ids,
	const data_type*data,
	const data_type*x,
	data_type*y)
{
	unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < n_rows)
	{
		data_type dot = 0;
		for (unsigned int element = 0; element < elements_in_rows; element++)
		{
			const unsigned int element_offset = row + element * n_rows;
			dot += data[element_offset] * x[col_ids[element_offset]];
		}
		y[row] = dot;
	}
}

template <typename data_type>
__global__ void coo_spmv_kernel(
	unsigned int n_elements,
	const unsigned int *col_ids,
	const unsigned int *row_ids,
	const data_type*data,
	const data_type*x,
	data_type*y)
{
	unsigned int element = blockIdx.x * blockDim.x + threadIdx.x;

	if (element < n_elements)
	{
		const data_type dot = data[element] * x[col_ids[element]];
		atomicAdd(y + row_ids[element], dot);
	}
}

measurement_class gpu_ell_spmv(
	const ell_matrix_class<float> &matrix,
	resizable_gpu_memory<float> &A,
	resizable_gpu_memory<unsigned int> &col_ids,
	resizable_gpu_memory<float> &x,
	resizable_gpu_memory<float> &y,

	float*reusable_vector,
	const float*reference_y)
{
	auto &meta = matrix.meta;

	const size_t A_size = matrix.get_matrix_size();
	const size_t col_ids_size = A_size;
	const size_t x_size = matrix.meta.cols_count;
	const size_t y_size = matrix.meta.rows_count;

	A.resize(A_size);
	col_ids.resize(col_ids_size);
	x.resize(x_size);
	y.resize(y_size);

	hipMemcpy(A.get(), matrix.data.get(), A_size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(col_ids.get(), matrix.columns.get(), col_ids_size * sizeof(unsigned int), hipMemcpyHostToDevice);

	{
		dim3 block_size = dim3(512);
		dim3 grid_size{};

		grid_size.x = (x_size + block_size.x - 1) / block_size.x;
		fill_vector<<<grid_size, block_size>>> (x_size, x.get(), 1.0);

		grid_size.x = (y_size + block_size.x - 1) / block_size.x;
		fill_vector<<<grid_size, block_size>>> (y_size, y.get(), 0.0);
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipDeviceSynchronize();
	hipEventRecord(start);
	{
		dim3 block_size = dim3(256);
		dim3 grid_size{};

		grid_size.x = (meta.rows_count + block_size.x - 1) / block_size.x;

		ell_spmv_kernel<<<grid_size, block_size>>> (meta.rows_count, matrix.elements_in_rows, col_ids.get(), A.get(), x.get(), y.get());
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	hipMemcpy(reusable_vector, y.get(), y_size * sizeof(float), hipMemcpyDeviceToHost);

	compare_results(y_size, reusable_vector, reference_y);

	const double elapsed = milliseconds / 1000;

	const unsigned int n_elements = matrix.elements_in_rows * matrix.meta.rows_count;
	const size_t data_bytes = n_elements * sizeof(float);
	const size_t x_bytes = n_elements * sizeof(float);
	const size_t col_ids_bytes = n_elements * sizeof(unsigned int);
	const size_t y_bytes = matrix.meta.rows_count * sizeof(float);

	const size_t operations_count = n_elements * 2; // + and * per element

	return measurement_class(
		"GPU ELL",
		elapsed,
		data_bytes + x_bytes + col_ids_bytes + y_bytes,
		operations_count);
}

measurement_class gpu_coo_spmv(
	const coo_matrix_class<float> &matrix,
	resizable_gpu_memory<float> &A,
	resizable_gpu_memory<unsigned int> &col_ids,
	resizable_gpu_memory<unsigned int> &row_ids,
	resizable_gpu_memory<float> &x,
	resizable_gpu_memory<float> &y,

	float*reusable_vector,
	const float*reference_y)
{
	const size_t n_elements = matrix.get_matrix_size();
	const size_t x_size = matrix.meta.cols_count;
	const size_t y_size = matrix.meta.rows_count;

	A.resize(n_elements);
	col_ids.resize(n_elements);
	row_ids.resize(n_elements);
	x.resize(x_size);
	y.resize(y_size);

	hipMemcpy(A.get(), matrix.data.get(), n_elements * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(col_ids.get(), matrix.cols.get(), n_elements * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(row_ids.get(), matrix.rows.get(), n_elements * sizeof(unsigned int), hipMemcpyHostToDevice);

	{
		dim3 block_size = dim3(512);
		dim3 grid_size{};

		grid_size.x = (x_size + block_size.x - 1) / block_size.x;
		fill_vector<<<grid_size, block_size>>> (x_size, x.get(), 1.0);

		grid_size.x = (y_size + block_size.x - 1) / block_size.x;
		fill_vector<<<grid_size, block_size>>> (y_size, y.get(), 0.0);
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipDeviceSynchronize();
	hipEventRecord(start);
	{
		dim3 block_size = dim3(512);
		dim3 grid_size{};

		grid_size.x = (n_elements + block_size.x - 1) / block_size.x;

		coo_spmv_kernel<<<grid_size, block_size>>> (n_elements, col_ids.get(), row_ids.get(), A.get(), x.get(), y.get());
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	hipMemcpy(reusable_vector, y.get(), y_size * sizeof(float), hipMemcpyDeviceToHost);

	compare_results(y_size, reusable_vector, reference_y);

	const double elapsed = milliseconds / 1000;

	const size_t data_bytes = matrix.meta.non_zero_count * sizeof(float);
	const size_t x_bytes = matrix.meta.non_zero_count * sizeof(float);
	const size_t col_ids_bytes = matrix.meta.non_zero_count * sizeof(unsigned int);
	const size_t row_ids_bytes = matrix.meta.non_zero_count * sizeof(unsigned int);
	const size_t y_bytes = matrix.meta.non_zero_count * sizeof(float);

	const size_t operations_count = matrix.meta.non_zero_count * 2; // + and * per element
	return measurement_class(
		"GPU COO",
		elapsed,
		data_bytes + x_bytes + col_ids_bytes + row_ids_bytes + y_bytes,
		operations_count);
}